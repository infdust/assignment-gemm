#include <stdio.h>
#include "gemm.h"
#include "perf_test.h"
class GPUTimer
{
    hipEvent_t __begin;
    hipEvent_t __end;

public:
    GPUTimer()
    {
        hipEventCreate(&__begin);
        hipEventCreate(&__end);
    }
    void start()
    {
        hipDeviceSynchronize();
        hipEventRecord(__begin);
        hipEventQuery(__begin);
    }
    double stop()
    {
        auto err = hipDeviceSynchronize();
        hipEventRecord(__end);
        hipEventSynchronize(__end);
        if (err != hipSuccess)
        {
            printf("ERROR! %s:%s\n", hipGetErrorName(err), hipGetErrorString(err));
            return 0.0;
        }
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("ERROR! %s:%s\n", hipGetErrorName(err), hipGetErrorString(err));
            return 0.0;
        }
        float time = 0.0f;
        hipEventElapsedTime(&time, __begin, __end);
        printf("%gms\n", time);
        return time / 1000.0;
    }
    ~GPUTimer()
    {
        hipEventDestroy(__begin);
        hipEventDestroy(__end);
    }
};
template <size_t MN, size_t K>
using gemmtest = PerfTest<MatMul<MN, K, MN>, GPUTimer>;
#ifndef MACRO_MN
#define MACRO_MN 8192
#endif
#ifndef MACRO_K
#define MACRO_K 8192
#endif
int main()
{
#ifdef MACRO_CUTLASS
    gemmtest<MACRO_MN, MACRO_K>().run<200, 1000>(0).run<200, 1000>(1).run<200, 1000>(2);
#else 
    gemmtest<MACRO_MN, MACRO_K>().run<200, 1000>(0).run<200, 1000>(1);
#endif
    return 0;
}